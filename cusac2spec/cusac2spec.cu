#include "hip/hip_runtime.h"
/*
./cusac2spec -i ../result/sac.sacum -o ../result/spec.spec -w 8 -l 4 -m3 1/0.1/0.3/0.4/0.5/5 -wf 0.1/0.3/0.4/0.5 -ns 5
./cusac2spec -i ../result/smallsac -o ../result/smallspec -w 7200 -l 3600 -m3 60/0.016/0.020/0.067/0.083/20 -wf 0.0055/0.0066/0.400/0.454 -ns 30
*/

#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <dirent.h>
#include <sys/types.h>
#include <sys/stat.h>
#include "globalvar.cuh"
#include "ini.cu"
#include "sac.cu"
#include "spec.cu"
#include "fft.cu"

void f(char *sacpath)
{
    FILE *sacfile=fopen(sacpath,"rb");
    readsac(sacfile);//读入sac的头和数据
    fclose(sacfile);
    spec_name_change(specrootpath,sacpath);//生成spec的完整文件名
    FILE *specfile=fopen(specrootpath,"wb");
    specrootpath[specl]='\0';//把spec路径改回去
    writespechead(specfile);//写spec的头

    hipMemcpy(d_sac_r1,sacdata,n*sor,HD);//d_sac_r1保存原始数据
    ////////////////分段存储+rrr////////////////
    for (int i=step-1;i>=0;i--)
    {
        if (winn+lagn*i<=n)
            segsize[i]=winn;
        else
            segsize[i]=n-lagn*i;
        hipMemset(d_sac_r2+i*nfft,0,nfft*sor);
        hipMemcpy(d_sac_r2+i*nfft,d_sac_r1+i*lagn,segsize[i]*sor,DD);
        rrrcontrol(d_sac_r2+i*nfft,segsize[i],d_sac_r1+i*nfft);///rrr=rdc+rtr
        //这里和原代码不同
        //原代码中先对所有数据作rrr,再对分段数据作rrr
        //这里直接对分段数据作rrr
    }
    hipMemcpy(d_segsize,segsize,step*soi,HD);
    hipMemcpy(d_sac_r1,d_sac_r2,step*nfft*sor,DD);//d_sac_r1,d_sac_r2保存分段数据
    ////////////////第一次FFT////////////////
    hipfftExecR2C(planR2C,d_sac_r2,d_sac_c1);
    for (int i=0;i<step;i++)//每段数据压缩到nspec位
    {
        hipfftComplex t;
        hipMemcpy(&t,d_sac_c1+i*(nspec+1)+nspec,soc,DH);
        t.y=t.x;
        t.x=0;
        hipMemcpy(d_sac_c1+i*(nspec+1),&t,soc,HD);
    }
    //d_sac_r2保存分段频域数据
    ////////////////第一次滤波////////////////
    m31n=m31/df;
    m32n=m32/df;
    m33n=m33/df;
    m34n=m34/df;
    if (m34n>=nspec)
        m34n=nspec-1;
    cuGenW1<<<(nspec+maxth-1)/maxth,maxth>>>(d_sac_r3,nspec,m31n,m32n,m33n,m34n);
    //d_sac_r3保存权重
    cuds;
    cuFilter<<<(step*nspec+maxth-1)/maxth,maxth>>>(step,nspec,d_sac_c1,d_sac_r3);
    cuds;
    ////////////////第二次FFT////////////////
    for (int i=0;i<step;i++)//每段数据扩展到nspec+1位
    {
        hipfftComplex t;
        hipMemcpy(&t,d_sac_c1+i*(nspec+1),soc,DH);
        t.x=t.y;
        t.y=0;
        hipMemcpy(d_sac_c1+i*(nspec+1)+nspec,&t,soc,HD);
        t.x=0;
        hipMemcpy(d_sac_c1+i*(nspec+1),&t,soc,HD);
    }
    hipfftExecC2R(planC2R,d_sac_c1,d_sac_r2);
    cuDivn<<<all>>>(d_sac_r2,step,nfft);
    cuds;
    //d_sac_r2保存滤波后的分段时域数据
    ////////////////临域取平均////////////////
    cuPos<<<all>>>(d_sac_r2,step*nfft);//全部取绝对值
    cuds;
    hipMemcpy(d_sac_r3,d_sac_r2,step*nfft*sor,DD);
    cuSmooth<<<all>>>(d_sac_r2,d_sac_r3,step,nfft,d_segsize,m3wn);
    ////////////////第一次归一化+削平////////////////
    cuNormalr<<<all>>>(d_sac_r1,d_sac_r2,step,nfft,d_segsize);
    //这里d_sac_r1为刚分好段时的时域数据,d_sac_r2为平滑后的数据
    cuds;
    cuCutTop<<<all>>>(d_sac_r1,step*nfft,m3top);
    cuds;
    ////////////////频谱白化////////////////
    if (wf)
    {
        ////////////////第三次FFT////////////////
        hipfftExecR2C(planR2C,d_sac_r1,d_sac_c1);
        for (int i=0;i<step;i++)//这里没有执行rrr,频域中一些之前为0的点现在不为0,压缩方法不一样
        {
            hipfftComplex t1,t2;
            hipMemcpy(&t1,d_sac_c1+i*(nspec+1),soc,DH);
            hipMemcpy(&t2,d_sac_c1+i*(nspec+1)+nspec,soc,DH);
            t1.y=t2.x;
            hipMemcpy(d_sac_c1+i*(nspec+1),&t1,soc,HD);
        }
        ////////////////第二次归一化////////////////
        cuGenW2<<<(step*(nspec+1)+maxth-1)/maxth,maxth>>>(d_sac_r2,d_sac_c1,step,nspec+1);
        cuSmooth<<<(step*(nspec+1)+maxth-1)/maxth,maxth>>>(d_sac_r3,d_sac_r2,step,nspec+1,NULL,ns);
        cuNormalc<<<all>>>(d_sac_c1,d_sac_r3,step,nspec+1);
        //此时频域数据在d_sac_c1里
        ////////////////第二次滤波////////////////
        wf1n=wf1/df;
        wf2n=wf2/df;
        wf3n=wf3/df;
        wf4n=wf4/df;
        if (wf4n>=nspec)
            wf4n=nspec-1;
        cuGenW1<<<(nspec+maxth-1)/maxth,maxth>>>(d_sac_r2,nspec,wf1n,wf2n,wf3n,wf4n);
        cuds;
        cuFilter<<<(step*nspec+maxth-1)/maxth,maxth>>>(step,nspec,d_sac_c1,d_sac_r2);
        cuds;
    }
    for (int i=0;i<step;i++)
    {
        hipMemcpy(specdata,d_sac_c1+i*(nspec+1),nspec*soc,DH);
        fwrite(specdata,nspec*soc,1,specfile);
    }

    fclose(specfile);
}

void brousesac(char *sacpath)
{
    if (strcmp(sacpath+strlen(sacpath)-strlen(sactail),sactail)==0)//输入路径是个sacum文件
    {
        f(sacpath);
        return;
    }
    if (!isfolder(sacpath))//不是sacum文件,且不是文件夹
        return;
    DIR *currdir=opendir(sacpath);
    addslash(sacpath);
    struct dirent *ent;
    int sacl=strlen(sacpath);//当输入路径是文件夹时有效,记录当前文件夹的路径的长度
    while ((ent=readdir(currdir))!=NULL)//遍历当前文件夹里所有文件(夹)
    {
        if (ent->d_name[strlen(ent->d_name)-1]=='.')
            continue;
        strcat(sacpath,ent->d_name);//现在sacpath为准备访问的文件(夹)名
        if (isfolder(sacpath))
            brousesac(sacpath);//如果是文件夹,则递归
        else
            f(sacpath);
        sacpath[sacl]='\0';//最后把加上的"当前访问的文件(夹)名"去掉
    }
    closedir(currdir);
}

int main(int argc,char **argv)
{
    s=clock();
    ini(argc,argv);
    if (isfolder(specrootpath))
        addslash(specrootpath);
    //如果输入路径是一个文件,则只处理这一个文件
    //如果输入路径是一个文件夹,则处理包含的所有文件
    //如果输出路径是一个文件,则要求输入路径也是文件,并直接写到文件内(不考虑文件名)
    //如果输出路径是一个文件夹,则在文件夹内新建和输入文件同名的文件(后缀名不同)并输出
    if ((isfolder(sacrootpath)) && (!isfolder(specrootpath)))
        return 0;
    specl=strlen(specrootpath);
    brousesac(sacrootpath);
    t=clock();
    printf("time=%fs   filenum=%d   datanum=%d\n",(float)(t-s)/1000000,filenum,datanum);
}