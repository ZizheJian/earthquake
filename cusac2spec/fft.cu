#include "hip/hip_runtime.h"
#ifndef __FFT_CU__
#define __FFT_CU__

#include "globalvar.cuh"

__global__ void cuMuli(float *data,int n)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=n)
        return;
    data[id]=data[id]*id;
}

__global__ void cuSum(float *data,int n,int gap)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=n)
        return;
    if (id%gap==0)
    {
        if (id+gap/2<n)
        {
            data[id]+=data[id+gap/2];
        }
    }
}

__global__ void currr(float *data,int n,float a,float b)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=n)
        return;
    data[id]-=(a*id+b);
}

void sumcontrol(float *data,int n,int muli)//muli为是否要乘上i的开关
{
    if (muli)
    {
        cuMuli<<<(n+maxth-1)/maxth,maxth>>>(data,n);
        cuds;
    }
    for (int i=2;i<=2*n;i*=2)
    {
        cuSum<<<(n+maxth-1)/maxth,maxth>>>(data,n,i);
        cuds;
    }
}

void rrrcontrol(float *data,int n,float* buffer)
{
    //buffer为给这个函数分配的一个寄存器,输入输出都是data
    float sum,summuli;//sum=sigma(data[i]),summuli=sigma(data[i]*i)
    hipMemcpy(buffer,data,n*sor,DD);
    sumcontrol(buffer,n,0);
    hipMemcpy(&sum,buffer,sor,DH);
    hipMemcpy(buffer,data,n*sor,DD);
    sumcontrol(buffer,n,1);
    hipMemcpy(&summuli,buffer,sor,DH);
    float n2=(n-1)*n*(2*n-1)/6;
    float n1=(n-1)*n/2;
    float n0=n;
    float a=(summuli*n0-sum*n1)/(n0*n2-n1*n1);
    float b=(summuli*n1-sum*n2)/(n1*n1-n0*n2);
    currr<<<(n+maxth-1)/maxth,maxth>>>(data,n,a,b);
    cuds;
}

__global__ void cuGenW1(hipfftReal* d_w,int n,int f1,int f2,int f3,int f4)
{
    //生成第一次滤波用的权重数组
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=n)
        return;
    if (id<f1)
        d_w[id]=0;
    else
        if (id<f2)
            d_w[id]=(1-cos(PI*(id-f1)/(f2-f1)))/2;
        else
            if (id<f3)
                d_w[id]=1;
            else
                if (id<f4)
                    d_w[id]=(1+cos(PI*(id-f3)/(f4-f3)))/2;
                else
                    d_w[id]=0;
}

__global__ void cuFilter(int step,int n,hipfftComplex* c,hipfftReal* w)//c乘上w,结果存在c里
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=step*n)
        return;
    int ida=(id/n)*(n+1)+id%n;//频域数据的下标
    //因为之前将每个频域分段从nspec+1个complex压缩到nspec个
    //所以这里要跳过那个分段里第nspec个complex
    int idb=id%n;//权重的下标
    c[ida].x*=w[idb];
    c[ida].y*=w[idb];
}

__global__ void cuDivn(hipfftReal *r,int step,int n)//总计step*n个float,全部除以n
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=step*n)
        return;
    r[id]=r[id]/n;
}

__global__ void cuPos(hipfftReal *a,int n)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=n)
        return;
    if (a[id]<0)
        a[id]*=-1;
}

__global__ void cuSmooth(hipfftReal *b,hipfftReal *a,int step,int n,int* segsize,int w)
{
    //b输出,a输入,w平滑半径
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=step*n)
        return;
    b[id]=0;
    int n1=id/n;//第几个分段
    int n2=id%n;//分段内位置
    int s,t;//平滑范围的头尾
    if (segsize==NULL)//没有提供具体的分段大小,则认为每一段长n
    {
        s=(n2-w>=0)?(n2-w):0;
        t=(n2+w<n)?(n2+w):(n-1);
    }
    else
    {
        if (n2>=segsize[n1])
            return;
        s=(n2-w>=0)?(n2-w):0;
        t=(n2+w<segsize[n1])?(n2+w):(segsize[n1]-1);
    }
    for (int i=s;i<=t;i++)
        b[id]+=a[n1*n+i];
    b[id]/=(2*w+1);
}

__global__ void cuNormalr(hipfftReal *b,hipfftReal *a,int step,int n,int *segsize)
{
    //b原始数据,a平滑数据
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=step*n)
        return;
    int n1=id/n;
    int n2=id%n;
    if (n2<segsize[n1])
        if (a[id]<1e-30)
            b[id]*=1e30;
        else
            b[id]/=a[id];
}

__global__ void cuNormalc(hipfftComplex *b,hipfftReal *a,int step,int n)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=step*n)
        return;
    if (a[id]<1e-30)
    {
        b[id].x*=1e30;
        b[id].y*=1e30;
    }
    else
    {
        b[id].x/=a[id];
        b[id].y/=a[id];
    }
}

__global__ void cuCutTop(hipfftReal *a,int n,float m)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=n)
        return;
    if (a[id]>m)
        a[id]=m;
    if (a[id]<-m)
        a[id]=-m;
}

__global__ void cuGenW2(hipfftReal* b,hipfftComplex* a,int step,int n)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if (id>=step*n)
        return;
    int n1=id/n;
    int n2=id%n;
    if (n2==0)
        b[id]=abs(a[n1*n].x);
    if (n2==n-1)
        b[id]=abs(a[n1*n].y);
    if ((n2!=0) && (n2!=n-1))
        b[id]=sqrt(a[id].x*a[id].x+a[id].y*a[id].y);
}

#endif