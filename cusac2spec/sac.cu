#ifndef __SAC_CU__
#define __SAC_CU__

#include "sac.cuh"
#include "globalvar.cuh"

void readsac(FILE *sacfile)//读如sac的头和数据
{
    fread(&sachd,sizeof(SACHEAD),1,sacfile);
    filenum++;
    datanum+=sachd.npts;
    n=sachd.npts;
    dt=sachd.delta;
    winn=win/dt;
    lagn=lag/dt;
    lagrate=(float)(n-winn+lagn-1)/lagn;//和原代码不同,考虑到最后一小段
    step=lagrate+1;
    m3wn=m3w/dt;
    nfft=2;
    while (nfft<winn)
        nfft*=2;
    nspec=nfft/2;
    df=1/(dt*nfft);
    if (n>maxn)
    {
        maxn=n;
        if (sacdata!=NULL)
        {
            free(sacdata);
            sacdata=NULL;
        }
    }
    if (step>maxstep)
    {
        maxstep=step;
        if (segsize!=NULL)
        {
            free(segsize);
            segsize=NULL;
        }
        if (d_segsize!=NULL)
        {
            hipFree(d_segsize);
            d_segsize=NULL;
        }
        if (d_sac_r1!=NULL)
        {
            hipFree(d_sac_r1);
            d_sac_r1=NULL;
        }
        if (d_sac_r2!=NULL)
        {
            hipFree(d_sac_r2);
            d_sac_r2=NULL;
        }
        if (d_sac_r3!=NULL)
        {
            hipFree(d_sac_r3);
            d_sac_r3=NULL;
        }
        if (d_sac_c1!=NULL)
        {
            hipFree(d_sac_c1);
            d_sac_c1=NULL;
        }
        if (planflag)
        {
            hipfftDestroy(planR2C);
            hipfftDestroy(planC2R);
            planflag=0;
        }
    }
    if (nspec>maxnspec)
    {
        maxnspec=nspec;
        if (specdata!=NULL)
        {
            free(specdata);
            specdata=NULL;
        }
        if (d_sac_r1!=NULL)
        {
            hipFree(d_sac_r1);
            d_sac_r1=NULL;
        }
        if (d_sac_r2!=NULL)
        {
            hipFree(d_sac_r2);
            d_sac_r2=NULL;
        }
        if (d_sac_r3!=NULL)
        {
            hipFree(d_sac_r3);
            d_sac_r3=NULL;
        }
        if (d_sac_c1!=NULL)
        {
            hipFree(d_sac_c1);
            d_sac_c1=NULL;
        }
        if (planflag)
        {
            hipfftDestroy(planR2C);
            hipfftDestroy(planC2R);
            planflag=0;
        }
    }
    if (sacdata==NULL)
        sacdata=(float*)malloc(maxn*sor);
    if (segsize==NULL)
        segsize=(int*)malloc(maxstep*soi);
    if (specdata==NULL)
        specdata=(float*)malloc(maxnspec*soc);
    if (d_segsize==NULL)
        hipMalloc((void**)&d_segsize,maxstep*soi);
    if (d_sac_r1==NULL)
        hipMalloc((void**)&d_sac_r1,maxstep*maxnspec*2*sor);
    if (d_sac_r2==NULL)
        hipMalloc((void**)&d_sac_r2,maxstep*maxnspec*2*sor);
    if (d_sac_r3==NULL)
        hipMalloc((void**)&d_sac_r3,maxstep*maxnspec*2*sor);
    if (d_sac_c1==NULL)
        hipMalloc((void**)&d_sac_c1,maxstep*(maxnspec+1)*soc);
    if (!planflag)
    {
        rank=1;
        len[0]=nfft;
        inembed[0]=nfft;    inembed[1]=step;
        istride=1;
        idist=nfft;
        onembed[0]=nspec+1; onembed[1]=step;
        ostride=1;
        odist=nspec+1;
        batch=step;
        hipfftPlanMany(&planR2C,rank,len,inembed,istride,idist,onembed,ostride,odist,HIPFFT_R2C,batch);
        hipfftPlanMany(&planC2R,rank,len,onembed,ostride,odist,inembed,istride,idist,HIPFFT_C2R,batch);
        planflag=1;
    }
    fread(sacdata,n*sor,1,sacfile);
}

#endif