#ifndef __INI_CU__
#define __INI_CU__

#include "globalvar.cuh"

void usage()
{
    printf("sac2spec -i sac -o spec -w win -l lag -m3 halfwidth/wf1/wf2/wf3/wf4/cutN [-wf bf1/bf2/bf3/bf4] [-caz cmpaz -cinc cmpinc] [-ns nsmooth]\n");
    printf("-w  窗口宽度(7200s)\n");
    printf("-l  滑动距离(3600s)\n");
    printf("-m3 halfwidth/wf1/wf2/wf3/wf4/cutN: -m3 60/0.016/0.020/0.067/0.083/20\n");
    printf("-wf bf1/bf2/bf3/bf4:    -wf 0.0055/0.0066/0.400/0.454\n");
    printf("-caz -cinc  ???\n");
    printf("-ns 30\n");
}

void ini(int argc,char** argv)//获取maxth和输入指令
{
    filenum=0;
    datanum=0;
    hipGetDeviceProperties(&gpuProp,0);
    maxth=gpuProp.maxThreadsPerBlock;
    maxn=0;
    maxstep=0;
    maxnspec=0;
    sacdata=NULL;
    specdata=NULL;
    segsize=NULL;
    d_segsize=NULL;
    d_sac_r1=NULL;
    d_sac_r2=NULL;
    d_sac_r3=NULL;
    d_sac_c1=NULL;
    planflag=0;
    if (argc <= 1) 
    {
        usage(); 
        exit(-1); 
    }
    for (int i=1;i<argc;i++) 
    {
        if (strcmp(argv[i],"-i")==0)
        {
            strcpy(sacrootpath,argv[i+1]);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-o")==0)
        {
            strcpy(specrootpath,argv[i+1]);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-w")==0)
        {
            win=atof(argv[i+1]);
            i++;
            continue;
        } 
        if (strcmp(argv[i],"-l")==0)
        {
            lag=atof(argv[i+1]);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-m3")==0) 
        {  
            sscanf(argv[i+1],"%f/%f/%f/%f/%f/%f",&m3w,&m31,&m32,&m33,&m34,&m3top);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-wf")==0) 
        {
            wf=1;
            sscanf(argv[i+1],"%f/%f/%f/%f",&wf1,&wf2,&wf3,&wf4);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-caz")==0) 
        {
            cmpaz=atof(argv[i+1]);
            cmp++;
            i++;
            continue;
        }
        if (strcmp(argv[i],"-cinc")==0) 
        {
            cmpinc=atof(argv[i+1]);
            cmp++;
            i++;
            continue;
        }
        if (strcmp(argv[i],"-ns")==0) 
        {
            ns=atoi(argv[i+1]);
            i++;
            continue;
        }
    }
}

#endif