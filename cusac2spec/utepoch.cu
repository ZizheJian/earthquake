#include "hip/hip_runtime.h"
#ifndef __UTEPOCH_CU__
#define __UTEPOCH_CU__

#include <stdio.h>
#include "sac.cuh"

long long int time2utepoch(int year,int jday,int hour,int min,int sec,int msec,int usec)
{
    long long int kk=1000000;
    long long int k=1000;
    long long int utepoch;
    int a4=year/4-!(year & 3);
    int a100=a4/25;
    int a400=a100/4;
    int intervening_leap_days=a4-a100+a400-477;
    int days=(365*(year-1970)+intervening_leap_days+jday-1);
    utepoch=(60*(60*((long long int)24*days+hour)+min)+sec)*kk+msec*k+usec;
    return utepoch;
}

long long int sactime2utepoch(SACHEAD hd,char mark)
{
    long long int kk=1000000;
    long long int utepoch;
    float marktime=0;
    int year,jday,hour,min,sec,msec;
    year=hd.nzyear;
    jday=hd.nzjday;
    hour=hd.nzhour;
    min=hd.nzmin;
    sec=hd.nzsec;
    msec=hd.nzmsec;
    if( mark=='b')
        marktime = hd.b;
    else 
        if(mark=='e')
            marktime=hd.e;
    utepoch=time2utepoch(year,jday,hour,min,sec,msec,0)+(long long int)((double)marktime*kk+0.5);
    return(utepoch);
}

#endif