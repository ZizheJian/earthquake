#include "hip/hip_runtime.h"
#ifndef __CC_CU__
#define __CC_CU__

#include "fft.cu"

__global__ void mul(hipfftComplex *c1,hipfftComplex *c2,intx2 *mlist,int winw,int segmax,int spec)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int mi=id/(segmax*spec);
    int segi=(id/spec)%segmax;
    int speci=id%spec;
    int x=mlist[mi].x;
    int y=mlist[mi].y;
    if (mi>=winw)
        return;
    int xx=(x*segmax+segi)*spec+speci;
    int yy=(y*segmax+segi)*spec+speci;
    int zz=(mi*segmax+segi)*(spec+1)+speci;
    if (speci==0)
    {
        c2[zz].x=c1[xx].x*c1[yy].x;
        c2[zz].y=c1[xx].y*c1[yy].y;
    }
    else
    {
        c2[zz].x=(1-(speci%2)*2)*(c1[xx].x*c1[yy].x+c1[xx].y*c1[yy].y);
        c2[zz].y=(1-(speci%2)*2)*(c1[xx].x*c1[yy].y-c1[xx].y*c1[yy].x);
    }
}

__global__ void add(hipfftComplex *c2,specinf *speclist,intx2 *mlist,int winw,int segmax,int spec,int gap,int gapnum)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int mi=id/(gapnum*spec);
    int segi=((id/spec)%gapnum)*gap*2;
    int speci=id%spec;
    int x=mlist[mi].x;
    int y=mlist[mi].y;
    if (mi>=winw)
        return;
    if ((segi+gap>=speclist[x].head.seg) || (segi+gap>=speclist[y].head.seg))
        return;
    int xx=(mi*segmax+segi)*(spec+1)+speci;
    int yy=(mi*segmax+segi+gap)*(spec+1)+speci;
    c2[xx].x+=c2[yy].x;
    c2[xx].y+=c2[yy].y;
}

__global__ void shift(hipfftComplex *c2,int winw,int segmax,int spec)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int mi=id;
    if (mi>=winw)
        return;
    int xx=mi*segmax*(spec+1);
    int yy=mi*segmax*(spec+1)+spec;
    c2[yy].x=c2[xx].y;
    c2[yy].y=0;
    c2[xx].y=0;
}

__global__ void div(hipfftReal *r1,specinf *speclist,intx2 *mlist,int winw,int spec)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int mi=id/(spec*2);
    if (mi>=winw)
        return;
    int x=mlist[mi].x;
    int y=mlist[mi].y;
    int seg=speclist[x].head.seg;
    if (seg>speclist[y].head.seg)
        seg=speclist[y].head.seg;
    int a=2*spec*seg;
    r1[id]/=a;
}

void cc(hipfftComplex *source,hipfftComplex *station,hipfftComplex *swap,int spec,float dt,float *cc,int cchalfn,int stepidx)
{
    memcpy(swap,station,spec*soc);
    swap[0]=cmplx(swap[0].x*source[0].x,swap[0].y*source[0].y);
    float aa=-1;
    for (int j=1;j<spec;j++)
    {
        swap[j]=cmltp(swap[j],conjg(source[j]));
        swap[j]=dmltp(aa,swap[j]);
        aa=-aa;
    }
    fftr(swap,spec,-dt);
    int ncc=2*cchalfn+1;
    memcpy(cc,(float*)swap+spec-cchalfn,ncc*sizeof(float));
}

#endif