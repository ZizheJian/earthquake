#include "hip/hip_runtime.h"
#ifndef __FFT_CU__
#define __FFT_CU__

#include "complex.cuh"

void fft(hipfftComplex *a,int n,float dt)
{
    double pi=-PI;
    if (dt<0) 
        pi=PI;
    for (int m=n/2,j=0,i=1;i<n-1;i++)
    {
        int k;
        for (k=m;k<=j;k/=2)
            j-=k;
        j+=k;
        if(i<j) 
        {
            hipfftComplex t=a[i];
            a[i]=a[j];
            a[j]=t;
        }
    }
    for (int m=1,step=2;m<n;m=step,step*=2)
    {
        hipfftComplex u=One;
        hipfftComplex w=cmplx(cos(pi/m),sin(pi/m));
        for (int j=0;j<m;j++)
        {
            for (int i=j;i<n;i+=step) 
            {
	            int k=i+m;
	            hipfftComplex t=cmltp(a[k], u);
	            a[k]=cplus(a[i],cngtv(t));
	            a[i]=cplus(a[i],t);
            }
            u=cmltp(u,w);
        }
    }
    if (dt<0) 
        dt=-1/(n*dt);
    for (int i=0;i<n;i++) 
        a[i]=dmltp(dt,a[i]);
}

void fftr(hipfftComplex *x,int n,float dt)
{
    int n2=n/2;
    float delw=PI/n;
    hipfftComplex isg=IMAGE;
    if (dt>0)
    {
        delw=-delw;
        isg=cngtv(isg);
        fft(x,n,dt);
    }
    x[0]=cmplx(x[0].x+x[0].y,x[0].x-x[0].y);
    float w=delw;
    for (int i=1;i<n2;i++)
    {
        int j=n-i;
        hipfftComplex t=conjg(x[j]);
        hipfftComplex g=cplus(x[i],t);
        hipfftComplex h=cplus(x[i],cngtv(t));
        h=cmltp(cmplx(cos(w),sin(w)),h);
        x[i]=dmltp(0.5,cplus(g,cmltp(isg,h)));
        x[j]=dmltp(0.5,cplus(conjg(g),cmltp(isg,conjg(h))));
        w+=delw;
    }
    x[n2]=conjg(x[n2]);
    if (dt<0) 
    {
        x[0]=dmltp(0.5,x[0]);
        fft(x,n,dt);
    }
}

#endif