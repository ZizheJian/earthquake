#ifndef __INI_CU__
#define __INI_CU__

#include "globalvar.cuh"
#include "file.cu"

void usage()
{
    printf("correlation -i specpath -o sacpath -c halfCCLength [-n KCMPNM] [-x] [-m gpumemuse] [-d debug]\n");
    printf("-i specpath\n");
    printf("-o sacpath\n");
    printf("-c half of cclenth: -C 3600\n");
    printf("-n kcmpnm\n");
    printf("-x output symmetric NCF or not\n");
    printf("-m proportion of GPU memory available, default=0.5\n");
    printf("-d use the single-thread algorithm to verify the result, larger number = more info\n");
}

void ini(int argc,char** argv)
{

    hipGetDeviceProperties(&gpuProp,0);
    maxth=gpuProp.maxThreadsPerBlock;
    printf("maxthread=%d\n",maxth);
	hipMemGetInfo(&gpusize,&totgpusize);
	printf("gpusize=%dMB\n",(int)(gpusize/1048576));
    if (argc<=1) 
    {
        usage(); 
        exit(-1); 
    }
    for (int i=1;i<argc;i++) 
    {
        if (strcmp(argv[i],"-i")==0)
        {
            strcpy(specpath,argv[i+1]);
            addslash(specpath);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-o")==0)
        {
            strcpy(sacpath,argv[i+1]);
            addslash(sacpath);
            sacl=strlen(sacpath);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-c")==0)
        {
            cchalf=atof(argv[i+1]);
            i++;
            continue;
        } 
        if (strcmp(argv[i],"-n")==0)
        {
            setchn=1;
            strcpy(ncfchnname,argv[i+1]);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-x")==0)
        {
            symmetric=1;
            continue;
        }
        if (strcmp(argv[i],"-m")==0)
        {
            gpuuse=atof(argv[i+1]);
            i++;
            continue;
        }
        if (strcmp(argv[i],"-d")==0)
        {
            debug=atoi(argv[i+1]);
            i++;
            continue;
        }
    }
}

#endif