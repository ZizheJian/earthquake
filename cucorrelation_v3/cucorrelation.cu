#include "hip/hip_runtime.h"
/*
./cucorrelation -i ../result/smallspec -o ../result/smallcc -c 3600 -x -m 0.5 -d 2
*/

#include "globalvar.cuh"
#include "ini.cu"
#include "file.cu"
#include "cc.cu"
#include "gpumanage.cu"

int main(int argc,char **argv)
{
    s=time(NULL);
    ini(argc,argv);
    ////////////////读入文件名和头////////////////
    specfilenum=0;
    prescan(specpath,1);
    speclist=(specinf*)malloc(specfilenum*sizeof(specinf));
    specfilenum=0;
    prescan(specpath,2);
    printf("specfilenum=%d\n",specfilenum);
    ////////////////文件分为团////////////////
    for (int i=0;i<specfilenum;i++)//排序
        for (int j=i+1;j<specfilenum;j++)
            if (speccp(speclist[i],speclist[j])==1)
            {
                specinf t=speclist[i];
                speclist[i]=speclist[j];
                speclist[j]=t;
            }
    chimax=1;
    chinum=1;
    for (int i=1,j=1;i<=specfilenum;i++)//统计最大团和团数
        if (i!=specfilenum)
            if (speccp(speclist[i],speclist[i-1])==0)
                j++;
            else
            {
                if (chimax<j)
                    chimax=j;
                j=1;
                chinum++;
            }
        else
            if (chimax<j)
                chimax=j;
    chin=(intx2*)malloc(chinum*soi*2);
    chin[0].x=0;
    for (int i=1,j=0;i<specfilenum;i++)//计算团的两端位置
        if (speccp(speclist[i],speclist[i-1])!=0)
        {
            chin[j].y=i-1;
            j++;
            chin[j].x=i;
        }
    chin[chinum-1].y=specfilenum-1;
    printf("chinum=%d, chimax=%d\n",chinum,chimax);
    ////////////////确定单个文件大小////////////////
    specmax=0;
    segmax=0;
    for (int i=0;i<specfilenum;i++)
    {
        if (specmax<speclist[i].head.spec)
            specmax=speclist[i].head.spec;
        if (segmax<speclist[i].head.seg)
            segmax=speclist[i].head.seg;
    }
    printf("segmax=%d, specmax=%d\n",segmax,specmax);
    ////////////////确定窗口大小并分配////////////////
    a=seta();
    printf("a=%d\n",a);
    c1=(hipfftComplex*)malloc(chimax*segmax*specmax*soc);
    r1=(hipfftReal*)malloc(a*specmax*sor*2);
    mlist=(intx2*)malloc(a*soi*2);
    hipMalloc((void**)&d_c1,chimax*segmax*specmax*soc);
    hipMalloc((void**)&d_c2,a*segmax*(specmax+1)*soc);
    hipMalloc((void**)&d_r1,a*specmax*sor*2);
    hipMalloc((void**)&d_speclist,chimax*sospi);
    hipMalloc((void**)&d_mlist,a*soi*2);
    ////////////////cc////////////////
    for (int chii=0;chii<chinum;chii++)
    {
        int mx=1;
        int my=0;
        printf("chii=%d\n",chii);
        ////////////////读入////////////////
        int spec=speclist[chin[chii].x].head.spec;
        float dt=speclist[chin[chii].x].head.dt;
        int cchalfn=cchalf/dt;
        int chincur=chin[chii].y-chin[chii].x+1;
        for (int filei_a=chin[chii].x;filei_a<=chin[chii].y;filei_a++)
        {
            int filei_r=filei_a-chin[chii].x;
            FILE *specfile=fopen(speclist[filei_a].name,"rb");
            SPECHEAD trashhead;
            fread(&trashhead,sosph,1,specfile);
            for (int segi=0;segi<speclist[filei_a].head.seg;segi++)
                fread(c1+segmax*spec*filei_r+segi*spec,spec*soc,1,specfile);
            fclose(specfile);
        }
        hipMemcpy(d_c1,c1,chincur*segmax*spec*soc,HD);
        hipMemcpy(d_speclist,speclist+chin[chii].x,chincur*sospi,HD);
        for (int wini=0;wini<(chincur*(chincur-1)/2+a-1)/a;wini++)
        {
            ////////////////分配任务////////////////
            int wins=wini*a;
            int wint=min(wins+a-1,chincur*(chincur-1)/2);
            int winw=wint-wins+1;
            for (int mi=0;mi<winw;mi++)
            {
                mlist[mi].x=mx;
                mlist[mi].y=my;
                if (my==mx-1)
                {
                    my=0;
                    mx++;
                }
                else
                    my++;
            }
            printf("\twini=%d\n",wini);
            hipMemcpy(d_mlist,mlist,a*soi*2,HD);
            ////////////////乘////////////////
            blocknum=(winw*segmax*spec+maxth-1)/maxth;
            mul<<<blocknum,maxth>>>(d_c1,d_c2,d_mlist,winw,segmax,spec);
            cuds;
            ////////////////加////////////////
            for (int gap=1;gap<segmax;gap*=2)
            {
                int gapnum=(segmax+gap-1)/(2*gap);
                blocknum=(winw*gapnum*spec+maxth-1)/maxth;
                add<<<blocknum,maxth>>>(d_c2,d_speclist,d_mlist,winw,segmax,spec,gap,gapnum);
                cuds;
            }
            printf("\t\tadd complete\n");
            for (int conj=0;conj<1;conj++)
            {
                ////////////////共轭////////////////
                ////////////////切换复数存储格式////////////////
                blocknum=(winw+maxth-1)/maxth;
                shift<<<blocknum,maxth>>>(d_c2,winw,segmax,spec);
                cuds;
                printf("\t\tshift complete\n");
                ////////////////fft////////////////
                len[0]=spec*2;
                inembed[0]=spec*2;
                inembed[1]=winw;
                onembed[0]=spec+1;
                onembed[1]=winw;
                hipfftPlanMany(&plan,1,len,onembed,1,segmax*(spec+1),inembed,1,spec*2,HIPFFT_C2R,winw);
                hipfftExecC2R(plan,d_c2,d_r1);
                hipfftDestroy(plan);
                printf("\t\tfft complete\n");
                ////////////////除////////////////
                blocknum=(winw*spec*2+maxth-1)/maxth;
                div<<<blocknum,maxth>>>(d_r1,d_speclist,d_mlist,winw,spec);
                cuds;
                printf("\t\tdiv complete\n");
                ////////////////输出////////////////
                hipMemcpy(r1,d_r1,winw*spec*2*sor,DH);
                for (int mi=0;mi<winw;mi++)
                {   
                    int srci=chin[chii].x+mlist[mi].x;
                    int stai=chin[chii].x+mlist[mi].y;
                    for (int k=strlen(speclist[srci].name)-1;k>=0;k--)
                        if (speclist[srci].name[k]=='/')
                        {
                            strcpy(tpath,speclist[srci].name+k+1);
                            break;
                        }
                    tpath[strlen(tpath)-strlen(spectail)]='\0';
                    strcat(sacpath,tpath);
                    strcat(sacpath,"+");
                    for (int k=strlen(speclist[stai].name)-1;k>=0;k--)
                        if (speclist[stai].name[k]=='/')
                        {
                            strcpy(tpath,speclist[stai].name+k+1);
                            break;
                        }
                    tpath[strlen(tpath)-strlen(spectail)]='\0';
                    strcat(sacpath,tpath);
                    strcat(sacpath,sactail);
                    FILE *sacfile=fopen(sacpath,"wb");
                    if (sacfile==NULL)
                        printf("open failed, sac=%s\n",sacpath);
                    else
                    {
                        fwrite(r1+mi*spec*2+spec-cchalfn,(2*cchalfn+1)*sor,1,sacfile);
                        fclose(sacfile);
                    }
                    if (debug>=1)
                    {
                        hipfftComplex* swap=(hipfftComplex*)malloc(spec*soc);
                        float* finalcc=(float*)malloc((2*cchalfn+1)*sor);
                        float* ingcc=(float*)malloc((2*cchalfn+1)*sor);
                        memset(finalcc,0,(2*cchalfn+1)*sor);
                        for (int step=0;step<min(speclist[srci].head.seg,speclist[stai].head.seg);step++)
                        {
                            cc(c1+mlist[mi].x*segmax*spec+step*spec,c1+mlist[mi].y*segmax*spec+step*spec,swap,spec,dt,ingcc,cchalfn,step);
                            for (int p=0;p<2*cchalfn+1;p++)
                                finalcc[p]+=ingcc[p];
                        }
                        float diff=0;
                        float avg=0;
                        for (int p=0;p<2*cchalfn+1;p++)
                        {
                            finalcc[p]/=min(speclist[srci].head.seg,speclist[stai].head.seg);
                            diff=max(diff,abs(finalcc[p]-r1[mi*spec*2+spec-cchalfn+p]));
                            avg+=abs(r1[mi*spec*2+spec-cchalfn+p]);
                        }
                        if (debug>=2)
                        {
                            printf("multi thread:\n");
                            for (int p=0;p<2*cchalfn+1;p++)
                                printf("%f ",r1[mi*spec*2+spec-cchalfn+p]);
                            printf("\nsingle thread:\n");
                            for (int p=0;p<2*cchalfn+1;p++)
                                printf("%f ",finalcc[p]);
                            getchar();
                        }
                        avg/=(2*cchalfn+1);
                        printf("src=%d,  \tsta=%d,  \t%s: diff=%f, avg=%f\n",srci,stai,sacpath,diff,avg);
                        if (diff>=0.00001)
                            getchar();
                        free(swap);
                        free(finalcc);
                        free(ingcc);
                    }
                    sacpath[sacl]='\0';
                }
            }
        }
    }
    t=time(NULL);
    printf("total time=%ds\n",(int)(t-s));
}